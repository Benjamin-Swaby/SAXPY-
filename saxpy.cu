#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */


__global__ void fill(float *a , float x)
{
   int index =  blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;
   
   for(int i = index; i < N; i += stride)
   {
       a[i] = x;
   }
}

__global__ void saxpy(float *x, float *y, float *result)
{
   int index = threadIdx.x + blockIdx.x * blockDim.x;
   int stride = blockDim.x * gridDim.x;
   
   for(int i = index; i < N; i += stride)
   {
       result[i] = 2 * x[i] + y[i];
   }
}


hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}



#define multi 20
int main()
{
    float *x, *y, *result;
    int size = N * sizeof (int); // The total number of bytes per vector
    
    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props = getDetails(deviceId);
    

    hipMallocManaged(&result, size);
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);
    
    hipMemPrefetchAsync(result, size, deviceId);
    hipMemPrefetchAsync(x, size, deviceId);
    hipMemPrefetchAsync(y, size, deviceId);
	
    int threads_per_block = 512;
    printf("number of sms :%d \n", props.multiProcessorCount);
    int number_of_blocks = props.multiProcessorCount * multi;
	
	hipStream_t stream_result; hipStreamCreate(&stream_result);
	hipStream_t stream_x; hipStreamCreate(&stream_x);
	hipStream_t stream_y; hipStreamCreate(&stream_y);

    fill<<<threads_per_block,number_of_blocks, 0 , stream_result>>>(result, 0.0); //result
    fill<<<threads_per_block,number_of_blocks, 0 , stream_x>>>(x, 1.0); // array x 
    fill<<<threads_per_block,number_of_blocks, 0 , stream_y>>>(y, 2.0); // array y
	
	hipStreamDestroy(stream_result); hipStreamDestroy(stream_x); hipStreamDestroy(stream_y);	
    

    //error variables
    hipError_t addVectorsErr;
    hipError_t asyncErr;

    saxpy <<< number_of_blocks, threads_per_block >>> ( x, y, result );
    hipMemPrefetchAsync(result, size, hipCpuDeviceId);
    
	
	addVectorsErr = hipGetLastError();

    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    
    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("y[%d] = %f, ", i, result[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("y[%d] = %f, ", i, result[i]);
    printf ("\n");

    hipFree( result ); hipFree( x ); hipFree( y );
}
